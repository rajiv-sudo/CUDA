
#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel function to add two arrays
__global__ void addArrays(int *a, int *b, int *c, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    int size = 100; // Size of the arrays
    int a[size], b[size], c[size]; // Host arrays
    int *d_a, *d_b, *d_c; // Device arrays

    // Initialize arrays
    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Allocate memory on GPU
    hipMalloc((void **)&d_a, size * sizeof(int));
    hipMalloc((void **)&d_b, size * sizeof(int));
    hipMalloc((void **)&d_c, size * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;

    // Launch kernel
    addArrays<<<numBlocks, blockSize>>>(d_a, d_b, d_c, size);

    // Copy result from device to host
    hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // Print result
    printf("Result:\n");
    for (int i = 0; i < size; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
