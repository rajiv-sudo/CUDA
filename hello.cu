#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    cuda_hello<<<1,1>>>();
    hipDeviceSynchronize();
    hipDeviceReset(); 
    return 0;
}
